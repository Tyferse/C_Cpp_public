#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

#define N 16
#define BSIZE 16


__global__ void transpose(float *in, float *out) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N) {
        out[i + N*j] = in[j + N*i];
    }
}

int main() {
    float (*A)[N] = new float[N][N];
    float (*B)[N] = new float[N][N];

    cout << "Initial matrix A:" << endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            A[i][j] = (float)(i*N + j + 1);
            cout << A[i][j] << ' ';  
        }
        cout << endl;
    }

    float *dev_A, *dev_B;
    hipMalloc((void**)&dev_A, N*N * sizeof(float));
    hipMalloc((void**)&dev_B, N*N * sizeof(float));

    hipMemcpy(dev_A, A, N*N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    dim3 threads(N >= BSIZE ? BSIZE: N, N >= BSIZE ? BSIZE : N);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);
    cout << "\nThreads: " << threads.x << " " << threads.y << endl;
    cout << "Blocks: " << blocks.x << " " << blocks.y << endl;

    hipEventRecord(start, 0);

    transpose<<<blocks, threads>>>(dev_A, dev_B);

    hipMemcpy(B, dev_B, N*N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    cout << "Transposed matrix:" << endl;
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            cout << B[i][j] << ' ';
        }
        cout << endl;
    }

    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);

    cout << endl << "for " << elapsed << " miliseconds" << endl;

    delete[] A;
    delete[] B;

    hipFree(dev_A);
    hipFree(dev_B);

    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
