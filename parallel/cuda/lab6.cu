#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

#define N 4
#define BSIZE 2


__global__ void matMul(float *a, float *b, float *c) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Индекс начала первой подматрицы А, обрабатываемой блоком.
    int aBegin = N * BSIZE * by;
    int aEnd = aBegin + N - 1;
     
    // Шаг перебора подматриц А. 
    int aStep = BSIZE; 
    
    // Индекс первой подматрицы B обрабатываемой блоком. 
    int bBegin = BSIZE * bx;

    // Шаг перебора подматриц В, 
    int bStep = BSIZE * N; 
    float sum = 0.0; // Вычuсляемый элеменm С. 
  
    // Цuкл по 16*16 подматpицaм 
    for (int ia = aBegin, ib = bBegin; ia < aEnd; ia += aStep, ib += bStep)  { 
        // Очередная подматрица А в разделяемой памяти. 
        __shared__ float as[BSIZE][BSIZE]; 
        // Очередная подматрица В в разделяемой памяти. 
        __shared__ float bs[BSIZE][BSIZE];

        // Загрузuть по одному элементу из А и В в разделяемую nамять. 
        as[ty][tx] = a[ia + N*ty + tx];
        bs[ty][tx] = b[ib + N*ty + tx];
 
        // Дождаться, когда обе подматрицы будут nолностью загружены. 
        __syncthreads(); 

        // Вычuсляем нужный элеменm nроuзведенuя зааруженных подматриц.
        int ic = N*BSIZE * by + BSIZE * bx; 
        for (int k = 0; k < BSIZE; k++) {
            sum += as[ty][k] * bs[k][tx];

            // дождаться, пока все осmальные нumu блока закончаm вычuслять свои элеменmы. 
            __syncthreads();

            // Заnuсать результат.  
            c[ic + N*ty + tx] = sum; 
        }
    }
}

int main() {
    float *A = new float[N*N];
    float *B = new float[N*N];
    float *C = new float[N*N];

    cout << "Initial matrix A (B = 0.1 * A):" << endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) { 
            A[i*N + j] = (float)(i + j);
            B[i*N + j] = (i + j) * 0.1;
            cout << A[i*N + j] << " ";
        }

        cout << endl;
    }

    float *dev_A, *dev_B, *dev_C;
    hipMalloc((void**)&dev_A, N*N * sizeof(float));
    hipMalloc((void**)&dev_B, N*N * sizeof(float));
    hipMalloc((void**)&dev_C, N*N * sizeof(float));

    hipMemcpy(dev_A, A, N*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, N*N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    dim3 threads(N >= BSIZE ? BSIZE: N, N >= BSIZE ? BSIZE : N);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);
    cout << "\nThreads: " << threads.x << " " << threads.y << endl;
    cout << "Blocks: " << blocks.x << " " << blocks.y << endl;

    hipEventRecord(start, 0);

    matMul<<<blocks, threads>>>(dev_A, dev_B, dev_C);

    hipMemcpy(C, dev_C, N*N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    cout << "Matrix multiplication result:" << endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            cout << C[i*N + j] << ' ';
        }
        cout << endl;
    }

    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);

    cout << endl << "for " << elapsed << " miliseconds" << endl;

    delete[] A;
    delete[] B;
    delete[] C; 

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
