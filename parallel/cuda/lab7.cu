#include <iostream>
// #include <stdio.h>
// #include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;

#define EPS 1e-4
#define N 8
#define BSIZE 4


__global__ void integrateBodies(float3 *newPos, float3 *newVel, float3 *oldPos, float3 *oldVel, float dt){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    float3 pos = oldPos[idx];
    float3 f = make_float3(0.0, 0.0, 0.0);
    __shared__ float3 sp[BSIZE];

    for(int i = 0; i < N; i += BSIZE) {
        sp[tid] = oldPos[i + tid];
        __syncthreads();

        float3 r;
        // for (int j = 0; j < BSIZE; j++) {
        r.x = sp[tid].x - pos.x;
        r.y = sp[tid].y - pos.y;
        r.z = sp[tid].z - pos.z;

        float invDist = 1.0 / sqrtf(r.x * r.x + r.y * r.y + r.z * r.z + EPS*EPS);
        float s = invDist * invDist * invDist;

        f.x += r.x * s;
        f.y += r.y * s;
        f.z += r.z * s;
        // }

        __syncthreads();
    }

    float3 vel = oldVel[idx];

    vel.x += f.x * dt;
    vel.y += f.y * dt;
    vel.z += f.z * dt;

    pos.x += vel.x * dt;
    pos.y += vel.y * dt;
    pos.z += vel.x * dt;

    newPos[idx] = pos;
    newVel[idx] = vel;

    printf("%d %.4f %.4f %.4f\n", idx, newPos[idx].x, newPos[idx].y, newPos[idx].z);
}

void randomInit(float3 *a, int n) {
    for(int i = 0; i < n; ++i){
        a[i].x = 2 * (float)rand() / RAND_MAX - 1;
        a[i].y = 2 * (float)rand() / RAND_MAX - 1;
        a[i].z = 2 * (float)rand() / RAND_MAX - 1;
        cout << a[i].x << " " << a[i].y << " " << a[i].z << endl;
    }

    cout << endl;
}


int main(){
    float3 *p = new float3[N];
    float3 *v = new float3[N];

    float3 *p_dev[2] = {NULL, NULL};
    float3 *v_dev[2] = {NULL, NULL};

    randomInit(p, N);
    randomInit(v, N);

    hipMalloc((void **) &p_dev[0], N * sizeof(float3));
    hipMalloc((void **) &p_dev[1], N * sizeof(float3));
    hipMalloc((void **) &v_dev[0], N * sizeof(float3));
    hipMalloc((void **) &v_dev[1], N * sizeof(float3));

    hipMemcpy(p_dev[0], p, N * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(v_dev[0], v, N * sizeof(float3), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    int ind = 0;
    for(int i = 0; i < 90; i++, ind = (ind + 1) % 2) {
        cout << endl << i + 1 << " iteration" << endl;
        integrateBodies <<<N / BSIZE, BSIZE>>> (p_dev[(ind + 1) % 2], v_dev[(ind + 1) % 2], p_dev[ind], v_dev[ind], 1./30);
        hipDeviceSynchronize();
    }    

    hipMemcpy(p, p_dev[(ind + 1) % 2], N * sizeof(float3), hipMemcpyDeviceToHost);
    hipMemcpy(v, v_dev[(ind + 1) % 2], N * sizeof(float3), hipMemcpyDeviceToHost);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);

    cout << endl << "Evaluated for " << elapsed << " miliseconds" << endl;

    delete p, 
    delete v;

    hipFree(p_dev[0]);
    hipFree(p_dev[1]);
    hipFree(v_dev[0]);
    hipFree(v_dev[1]);

    return 0;
}
