#include <iostream>
#include <hip/hip_runtime.h>

#define N 16
#define M 12
#define BSIZE 16

using namespace std;


__global__ void MatScalar(float A[N][M], float B[N][M], float *result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    // printf("%d %d\n", idx, idy);
    if (idx < N && idy < M) {
        // printf("%d %d\n", idx, idy);
        atomicAdd(result, A[idx][idy] * B[idx][idy]);
    }
}


int main() {
		float (*A)[M] = new float[N][M];
    float (*B)[M] = new float[N][M];
  	float result = 0.0f;

    // cout << "Maxrix A:\n";
  	for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            A[i][j] = (float)(i * j); 
            B[i][j] = (float)(i * j) / 10.;
            // cout << A[i][j] << " ";
        }
        // cout << endl;
    }

    // cout << "\nMatrix B:\n";
    // for (int i = 0; i < N; ++i) {
    //     for (int j = 0; j < M; ++j) {
    //         cout << B[i][j] << " ";
    //     }
    //     cout << endl;
    // }
    
    float (*dev_A)[M], (*dev_B)[M], *dev_result;
	  hipMalloc((void**)&dev_A, sizeof(float) * N * M);
	  hipMalloc((void**)&dev_B, sizeof(float) * N * M);
    hipMalloc((void**)&dev_result, sizeof(float));
 
    hipMemcpy(dev_A, A, sizeof(float) * N * M, hipMemcpyHostToDevice);
	  hipMemcpy(dev_B, B, sizeof(float) * N * M, hipMemcpyHostToDevice);
    hipMemcpy(dev_result, &result, sizeof(float), hipMemcpyHostToDevice);


    dim3 threadsPerBlock(N >= BSIZE ? BSIZE: N, M >= BSIZE ? BSIZE : N);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    cout << "\nThreads: " << threadsPerBlock.x << " " << threadsPerBlock.y << endl;
    cout << "Blocks: " << numBlocks.x << " " << numBlocks.y << endl;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    MatScalar<<<numBlocks, threadsPerBlock>>>(dev_A, dev_B, dev_result);
		// cudaDeviceSynchronize();

	  hipMemcpy(&result, dev_result, sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);

    cout << endl << result << " for " << elapsed << " miliseconds";
  
    delete[] A;
    delete[] B;
    delete &result;
  
    hipFree(dev_A);
	  hipFree(dev_B);
	  hipFree(&dev_result);

    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
