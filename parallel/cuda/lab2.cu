#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


__global__ void kernel ( void ) 
{
  int ID  = blockIdx.x * blockDim.x + threadIdx.x;
    // blockIdx.x номер блока
    //blockDim.x количество потоков в блоке
    //threadIdx.x номер потока в блоке
  
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    printf("identificator, or number of thread is %d\n",ID); 
}


int main() {
    cout << "Hello from CPU!" << endl;
  
    kernel<<< 10, 1 >>>(); //gpu 

    hipDeviceSynchronize();
    return 0;
}
