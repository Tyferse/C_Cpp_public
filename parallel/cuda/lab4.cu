#include <iostream>
#include <hip/hip_runtime.h>

#define N 16
#define M 12
#define BSIZE 256

using namespace std;


__global__ void VecMul(float A[N*M], float B[N*M], float C[N*M]) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("%d %d\n", idx, idy);
    if (idx < N*M) {
        // printf("%d %d\n", idx, idy);
        C[idx] = A[idx] * B[idx];
    }
}

__global__ void reduction(float in[N*M], float *out) {
    extern __shared__ float sdata[];

    // Индекс текущего потока
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Загружаем данные в shared memory
    sdata[tid] = (i < N*M) ? in[i] : 0; 
    __syncthreads();

    // Редукция
    for (int s = 1; s < blockDim.x; s *= 2) {
        if ((tid % (2*s)) == 0)
            sdata[tid] += sdata[tid + s];
        
        __syncthreads();
    }

    // Записываем результат в output
    if (tid == 0) {
        out[blockIdx.x] = sdata[0];
    }
}


int main() {
	float *A = new float[N*M];
    float *B = new float[N*M];
    // float *result = new float[N*M];

    // cout << "Maxrix A:\n";
  	for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            A[i*M + j] = (float)(i * j);  
            B[i*M + j] = (float)(i * j) * 0.1;
            // cout << A[i*N + j] << " ";
        }
        // cout << endl;
    }

    // cout << "\nMatrix B:\n";
    // for (int i = 0; i < N; ++i) {
    //     for (int j = 0; j < M; ++j) {
    //         cout << B[i*N + j] << " ";
    //     }
    //     cout << endl;
    // }
    
    float *dev_A, *dev_B, *dev_result;
	hipMalloc((void**)&dev_A, sizeof(float) * N * M);
	hipMalloc((void**)&dev_B, sizeof(float) * N * M);
    hipMalloc((void**)&dev_result, sizeof(float) * N * M);

    hipMemcpy(dev_A, A, sizeof(float) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, sizeof(float) * N * M, hipMemcpyHostToDevice);
    // cudaMemcpy(dev_result, &result, sizeof(float) * N * M, cudaMemcpyHostToDevice);

    int blocks = (N*M + BSIZE - 1) / BSIZE;
    cout << "\nThreads: " << BSIZE << endl;
    cout << "Blocks: " << blocks << endl;

    float *reduced = new float[blocks], *dev_reduced;
    hipMalloc((void**)&dev_reduced, sizeof(float) * blocks);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    VecMul<<<blocks, BSIZE>>>(dev_A, dev_B, dev_result);
	hipDeviceSynchronize();

    reduction<<<blocks, BSIZE, BSIZE * sizeof(float)>>>(dev_result, dev_reduced);
    hipDeviceSynchronize();

    hipMemcpy(reduced, dev_reduced, sizeof(float) * blocks, hipMemcpyDeviceToHost);

    float res_sum = 0.;
    for (int i = 0; i < blocks; i++) {
        cout << reduced[i] << endl;
        res_sum += reduced[i];
    }

    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);

    cout << endl << res_sum << " for " << elapsed << " miliseconds";
    

    delete[] A;
    delete[] B;
    delete[] reduced;
    delete &res_sum;
  
    hipFree(dev_A);
	hipFree(dev_B);
	hipFree(&dev_result);
    hipFree(&dev_reduced);

    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
